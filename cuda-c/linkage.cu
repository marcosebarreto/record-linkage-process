// Linkage Process using CUDA C API
// Authors: Clicia Santos Pinto and Pedro Marcelino Mendes Novaes Melo

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <omp.h>

#define NCOL 101

__device__ int contador = 0;

void fill_matrix(int *matrix, int pos, char *line);
int get_num_of_lines(FILE *fp);
void process_file(FILE *fp, int *matrix);
void print_matrix(int *matrix, int nlines);
__global__ void kernel(int *matrixA, int *matrixB, int nlines_a, int nlines_b);
__device__ float dice(int *bloomA, int *bloomB);


int main(int argc, char const *argv[]) {
    double t1, t2;
    t1 = omp_get_wtime();

    FILE *base_a, *base_b;
    char file1[30];
    strcpy(file1, "base_");
    strcat(file1, argv[2]);
    strcat(file1, "K.bloom");
    // strcpy(file1, argv[1]);
    // strcpy(file2, argv[2]);

    int nlines_a = 0, nlines_b = 0;
    // int threads_per_block = 16;
    int threads_per_block = atoi(argv[1]);

    // opening large base (base_a) and small base (base_b)
    // printf("[LOADING DATABASES ... ]\n");
    base_a = fopen(file1, "r");
    base_b = fopen("base_1000K.bloom", "r");

    // --------------------- OPERATIONS WITH BASE A --------------------- //
    // getting line quantity
    // printf("[GETTING NUMBER LINES FOR BASE A ... ]\n");
    nlines_a = get_num_of_lines(base_a);
    int *matrixA = (int *)malloc(nlines_a * NCOL * sizeof(int));

    // processing base_a to fill matrixA
    // printf("[PROCESSING BASE A ... ]\n");
    process_file(base_a, matrixA);
    // print_matrix(matrixA, nlines_a);

    // --------------------- OPERATIONS WITH BASE B --------------------- //
    // getting line quantity
    // printf("[GETTING NUMBER LINES FOR BASE B ... ]\n");
    nlines_b = get_num_of_lines(base_b);
    int *matrixB = (int *)malloc(nlines_b * NCOL * sizeof(int));

    // processing base_b to fill matrixB
    // printf("[PROCESSING BASE B ... ]\n");
    process_file(base_b, matrixB);
    // print_matrix(matrixB, nlines_b);

    // printf("line_a=%d\tline_b=%d\n", nlines_a, nlines_b);

    // ------------------------- CUDA OPERATIONS ------------------------ //
    int *matrixA_d, *matrixB_d;

    // allocating device memory using a cuda function
    hipMalloc((int **)&matrixA_d, nlines_a * NCOL * sizeof(int));
    hipMalloc((int **)&matrixB_d, nlines_b * NCOL * sizeof(int));

    // copying host memory to device
    hipMemcpy(matrixA_d, matrixA, nlines_a * NCOL * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(matrixB_d, matrixB, nlines_b * NCOL * sizeof(int), hipMemcpyHostToDevice);

    // kernel operations
    // printf("[OPERATING AT KERNEL CUDA ... ]\n");
    dim3 dimGrid = (int) ceil( (int) nlines_a / (int) threads_per_block);
    dim3 dimBlock = threads_per_block;
    kernel<<<dimGrid, dimBlock>>>(matrixA_d, matrixB_d, nlines_a, nlines_b);

    hipDeviceSynchronize();

    // deallocating device memory
    hipFree(matrixA_d);
    hipFree(matrixB_d);

    free(matrixA);
    free(matrixB);

    // close files
    fclose(base_a);
    fclose(base_b);

    t2 = omp_get_wtime();

    // printf("%d\t%d\t%d\t%f\n", nlines_a, threads_per_block, nlines_a/threads_per_block, t2-t1);
    int tam_prob = atoi(argv[2]);
    printf("%d\t%f\n", (tam_prob * 1000), (t2-t1));

    return 0;
}


// function to get the number of lines of the file
int get_num_of_lines(FILE *fp) {
    int lines = 0;
    char line[256];
    if(!fgets(line, 255, fp))
        printf("fp = NULL");

    while (!feof(fp)) {
        lines++;
        if(!fgets(line, 255, fp))
            break;
    }

    return lines;
}


// function to get line by line of the file
void process_file(FILE *fp, int *matrix) {
    char line[256];
    int pos_to_insert = 0;

    rewind(fp);

    // getting line by line to insert into matrix
    if(!fgets(line, 255, fp))
        printf("fp = NULL");
    while (!feof(fp)) {
        line[strlen(line) - 1] = '\0';
        fill_matrix(matrix, pos_to_insert, line);

        pos_to_insert++;
        if(!fgets(line, 255, fp))
            break;
    }
}


// function to split a line and to insert the elements in matrix
void fill_matrix(int *matrix, int pos, char *line) {
    int i = 0, j = 0;
    char c, id[10];

    do {
        c = line[j];
        id[j] = c;
        j++;
    } while (c != ';');
    id[j-1] = '\0';
    // printf("ncol * pos: %d\n", NCOL * pos);
    matrix[NCOL * pos] = atoi(id);

    for (i = 1; i < NCOL; i++) {
        matrix[NCOL * pos + i] = line[j] - '0';
        j++;
    }
}


void print_matrix(int *matrix, int nlines) {
    int i, j;

    // for (i = 0; i < NCOL * nlines; i += 101) {
    //     printf("%d | ", matrix[i]);
    // }
    // printf("\n");

    for (i = 0; i < nlines; i++) {
        for (j = 0; j < NCOL; j++) {
            printf("%d", matrix[i * NCOL + j]);
        }
        printf("\n");
    }
    printf("\n");
}


__global__ void kernel(int *matrixA, int *matrixB, int nlines_a, int nlines_b){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("I = %d - blockID.x= %d e blockId.y = %d -- blockDim.x = %d e blockDim.y = %d -- threadIdx.x = %d e threadIdx.y = %d\n", i, blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, threadIdx.x, threadIdx.y);
    // int j = blockIdx.y * blockDim.y + threadIdx.y;

    int bloomA[100], bloomB[100];

    if (i < nlines_a) {
        // printf("%d ", matrixA[i * NCOL]);

        // getting bloom filter for each matrixA register
        for (int j = 1; j < 101; j++) {
            bloomA[j - 1] = matrixA[i * NCOL + j];
        }

        // getting bloom filter for each matrixB register
        for (int k = 0; k < nlines_b; k++) {
            for (int l = 1; l < 101; l++) {
                bloomB[l - 1] = matrixB[k * NCOL + l];
            }
            dice(bloomA, bloomB);
        }
    }

    // printf("num de comparacoes para thread %d: %d\n", i, contador);
}


// device function to calculate dice coefficient using bloom filter
__device__ float dice(int *bloomA, int *bloomB) {
    float a = 0, b = 0, h = 0;
    int i;

    for (i = 0; i < 100; i++) {
        if (bloomA[i] == 1) {
            a++;
            if (bloomB[i] == 1)
                h++;
        }
        if (bloomB[i] == 1) {
            b++;
        }
    }
    float dice = ((h * 2.0) / (a + b)) * 10000;
    // printf("%.1f\n", dice);
    // contador++;

    return dice;
}
